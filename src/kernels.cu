#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <hiprand/hiprand_kernel.h>
#include "tensor.h"
#include "kernels.h"


template<typename T>
__global__ void addT(const T* a, const T* b, T* result, size_t size){
	size_t index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < size){
		result[index] = a[index] + b[index];
	}
}

template<typename T>
__global__ void subT(const T* a, const T* b, T* result, size_t size){
	size_t index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < size){
		result[index] = a[index] - b[index];
	}
}


template<typename T>
__global__ void divT(const T* a, const T* b, T* result, size_t size){
	size_t index = threadIdx.x + blockDim.x * blockIdx.x;
	if(index < size){
		result[index] = a[index] / b[index];
	}
}

template<typename T>
__global__ void addS(const T* a, const float scalar, T* result, size_t size){
	size_t index = threadIdx.x + blockDim.x * blockIdx.x;
	if(index < size){
		result[index] = a[index] + scalar;
	}
}


template<typename T>
__global__ void subS(const T* a, const float scalar, T* result, size_t size){
	size_t index = threadIdx.x + blockDim.x * blockIdx.x;
	if(index < size){
		result[index] = a[index] - scalar;
	}
}


template<typename T>
__global__ void mulS(const T* a, const float scalar, T* result, size_t size){
	size_t index = threadIdx.x + blockDim.x * blockIdx.x;
	if(index < size){
		result[index] = a[index] * scalar;
	}
}


template<typename T>
__global__ void divS(const T* a, const float scalar, T* result, size_t size){
	size_t index = threadIdx.x + blockDim.x * blockIdx.x;
	if(index < size){
		result[index] = a[index] / scalar;
	}
}


template<typename T>
void addTwoTensors(const Tensor<T>& tensor1, const Tensor<T>& tensor2, Tensor<T>& tensor3){
	size_t size1 = tensor1.size();
	size_t size2 = tensor2.size();

	size_t size3 = tensor3.size();
	if(size1 != size2 && size1 != size3){
		throw std::runtime_error("Invalid sizes for add tensors");
	}

	dim3 tpb(16, 16);
	dim3 bpg((size3 + tpb.x - 1)/tpb.x, (size3 + tpb.y - 1) / tpb.y);

	addT<<<bpg, tpb>>>(tensor1.device_data(), tensor2.device_data(), tensor3.device_data(), size3);

	hipError_t err = hipGetLastError();
	if(err != hipSuccess){
		throw std::runtime_error("CUDA addTensors kernel launch failed.");
	}
}

template<typename T>
void subTwoTensors(const Tensor<T>& tensor1, const Tensor<T>& tensor2, Tensor<T>& tensor3){
	size_t size1 = tensor1.size();
	size_t size2 = tensor2.size();
	size_t size3 = tensor3.size();
	if(size1 != size2 && size1 != size3){
		throw std::runtime_error("Invalid sizes for subtract tensors");
	}
	dim3 tpb(16, 16);
	dim3 bpg((size3 + tpb.x - 1) / tpb.x, (size3 + tpb.y - 1) / tpb.y);

	
	subT<<<bpg, tpb>>>(tensor1.device_data(), tensor2.device_data(), tensor3.device_data(), size3);
	
	hipError_t err = hipGetLastError();
	if(err != hipSuccess){
		throw std::runtime_error("CUDA subTensors kernel launch failed.");
	}
}


template<typename T>
__global__ void mulT(const T* a, const T* b, T* result, size_t m, size_t n, size_t k){
	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	if(row < m && col < n){
		T dotProd = 0;
		for(int i = 0; i < k; i++){
			dotProd += a[row * k + i] * b[i * n + col];
		}
		result[row * n + col] = dotProd;
	}
}


template<typename T>
__global__ void addTV(const T* a, const T* b, T* result, size_t m, size_t n, size_t k){
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
		
	if(row < m && col < k){
		result[row * k + col] = a[row * k + col] + b[row];
	}
}

template<typename T>
void addTensorAndVector(const Tensor<T>& tensor1, const Tensor<T>& vector1, Tensor<T>& tensor3){
	size_t m, k, n;
	m = tensor1.shape()[0];
	k = tensor1.shape()[1];
	n = vector1.shape()[1];
	if(n != 1) throw std::runtime_error("Matrix-vector adding broadcasting issue.");
	
	if(m != tensor3.shape()[0] || k != tensor3.shape()[1]) throw std::runtime_error("Result tensor shape mismatch.");

	dim3 tpb(16, 16);
	dim3 bpg((k + tpb.x - 1) / tpb.x, (m + tpb.y - 1) / tpb.y);
	addTV<<<bpg, tpb>>>(tensor1.device_data(), vector1.device_data(), tensor3.device_data(), m, n, k);
	hipError_t err = hipGetLastError();
	if(err != hipSuccess){
		throw std::runtime_error("CUDA tensor x vector failed.");
	}
}


template<typename T>
void mulTwoTensors(const Tensor<T>& tensor1, const Tensor<T>& tensor2, Tensor<T>& tensor3){
	size_t m, k, n;
	m = tensor1.shape()[0];
	n = tensor2.shape()[1];
	k = tensor1.shape()[1];
	
	if(k != tensor2.shape()[0]) throw std::runtime_error("Matrix dim do not match - k value.");
	if(m != tensor3.shape()[0] || n != tensor3.shape()[1]) throw std::runtime_error("Result matrix dims are off");
	dim3 tpb(16, 16);
	dim3 bpg((n + tpb.x - 1) / tpb.x, (m + tpb.y - 1) / tpb.y);

	mulT<<<bpg, tpb>>>(tensor1.device_data(), tensor2.device_data(), tensor3.device_data(), m, n, k);
	
	hipError_t err = hipGetLastError();
	if(err != hipSuccess){
		throw std::runtime_error("CUDA subTensors kernel launch failed.");
	}
}


template<typename T>
void divTwoTensors(const Tensor<T>& tensor1, const Tensor<T>& tensor2, Tensor<T>& tensor3){
	size_t size1 = tensor1.size();
	size_t size2 = tensor2.size();
	size_t size3 = tensor3.size();
	if(size1 != size2 && size1 != size3){
		throw std::runtime_error("Invalid sizes for subtract tensors");
	}
	dim3 tpb(16, 16);
	dim3 bpg((size3 + tpb.x - 1) / tpb.x, (size3 + tpb.y - 1) / tpb.y);

	
	divT<<<bpg, tpb>>>(tensor1.device_data(), tensor2.device_data(), tensor3.device_data(), size3);
	
	hipError_t err = hipGetLastError();
	if(err != hipSuccess){
		throw std::runtime_error("CUDA subTensors kernel launch failed.");
	}
}

template<typename T>
void addScalar(const Tensor<T>& tensor1, Tensor<T>& tensor3, const float scalar){
	size_t size1 = tensor1.size();
	size_t size3 = tensor3.size();
	if(size1 != size3){
		throw std::runtime_error("Invalid sizes for add scalar.");
	}
	dim3 tpb(16, 16);
	dim3 bpg((size3 + tpb.x - 1) / tpb.x, (size3 + tpb.y - 1) / tpb.y);
	
	addS<<<bpg, tpb>>>(tensor1.device_data(), scalar, tensor3.device_data(), size1);
	hipError_t err = hipGetLastError();
	if(err != hipSuccess){
		throw std::runtime_error("CUDA add scalar kernel launch failed.");
	}
}	


template<typename T>
void subScalar(const Tensor<T>& tensor1, Tensor<T>& tensor3, const float scalar){
	size_t size1 = tensor1.size();
	size_t size3 = tensor3.size();
	if(size1 != size3){
		throw std::runtime_error("Invalid sizes for add scalar.");
	}
	dim3 tpb(16, 16);
	dim3 bpg((size3 + tpb.x - 1) / tpb.x, (size3 + tpb.y - 1) / tpb.y);
	
	subS<<<bpg, tpb>>>(tensor1.device_data(), scalar, tensor3.device_data(), size1);
	hipError_t err = hipGetLastError();
	if(err != hipSuccess){
		throw std::runtime_error("CUDA add scalar kernel launch failed.");
	}
}	


template<typename T>
void mulScalar(const Tensor<T>& tensor1, Tensor<T>& tensor3, const float scalar){
	size_t size1 = tensor1.size();
	size_t size3 = tensor3.size();
	if(size1 != size3){
		throw std::runtime_error("Invalid sizes for add scalar.");
	}
	dim3 tpb(16, 16);
	dim3 bpg((size3 + tpb.x - 1) / tpb.x, (size3 + tpb.y - 1) / tpb.y);
	
	mulS<<<bpg, tpb>>>(tensor1.device_data(), scalar, tensor3.device_data(), size1);
	hipError_t err = hipGetLastError();
	if(err != hipSuccess){
		throw std::runtime_error("CUDA add scalar kernel launch failed.");
	}
}	


template<typename T>
void divScalar(const Tensor<T>& tensor1, Tensor<T>& tensor3, const float scalar){
	size_t size1 = tensor1.size();
	size_t size3 = tensor3.size();
	if(size1 != size3){
		throw std::runtime_error("Invalid sizes for add scalar.");
	}
	dim3 tpb(16, 16);
	dim3 bpg((size3 + tpb.x - 1) / tpb.x, (size3 + tpb.y - 1) / tpb.y);
	
	divS<<<bpg, tpb>>>(tensor1.device_data(), scalar, tensor3.device_data(), size1);
	hipError_t err = hipGetLastError();
	if(err != hipSuccess){
		throw std::runtime_error("CUDA add scalar kernel launch failed.");
	}
}	

template<typename T>
__global__ void randN(T *a, const size_t size, unsigned long long seed){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < size){
		hiprandState state;
		hiprand_init(seed, index, 0, &state);
		a[index] = hiprand_normal(&state) * sqrtf(2.f/(size/2));
	}
}

template<typename T>
void fillRandom(Tensor<T>& tensor1, const size_t size){
	dim3 tpb(256);
	dim3 bpg((size + tpb.x - 1) / tpb.x);

	randN<<<bpg, tpb>>>(tensor1.device_data(), size, time(NULL));
	hipError_t err = hipGetLastError();
	if(err != hipSuccess){
		throw std::runtime_error("CUDA randN kernel launch failed.");
	}
}

template<typename T>
__global__ void forwardP(const T* in, const T* weights, const T* bias, T *out, const size_t m, const size_t k, const size_t n){
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	if(row < m && col < n){
		T dotProd = 0.f;
		for(size_t i = 0; i < k; i++){
			dotProd += in[row * k + i] * weights[n * i + col];
		}
		out[row * n + col] = dotProd + bias[row];
	}
}

template<typename T>
__global__ void relu(const T* in, T* out, const size_t m, const size_t n){
	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;

	if(row < m && col < n){
		if(in[row * n + col] <= 0) out[row * n + col] = 0;
		else out[row * n + col] = in[row * n + col];
	}
}

template<typename T>
void forwardCall(const Tensor<T>& in, const Tensor<T>& weights, const Tensor<T>& bias, Tensor<T>& out){
	size_t in_m, in_k;
	size_t w_k, w_n;
	in_m = in.shape()[0];
	in_k = in.shape()[1];
	w_k = weights.shape()[0];
	w_n = weights.shape()[1];
	
	size_t bias_m = bias.shape()[0];
	size_t bias_k = bias.shape()[1];

	dim3 tpb(16, 16);
	dim3 bpg( (w_n + tpb.x - 1) / tpb.x, (in_m + tpb.y - 1) / tpb.y);

	forwardP<<<bpg, tpb>>>(in.device_data(), weights.device_data(), bias.device_data(), out.device_data(), in_m, in_k, w_n);
	hipError_t err = hipGetLastError();
	if(err != hipSuccess){
		std::cerr << "CUDA ERR: " << hipGetErrorString(err) << std::endl;
		throw std::runtime_error("forward kernel failed");
	}
}

template<typename T>
void reluCall(const Tensor<T>& in, Tensor<T>& out){
	const size_t in_m = in.shape()[0];
	const size_t in_n = in.shape()[1];
	
	dim3 tpb(16,16);
	dim3 bpg((in_n + tpb.x - 1) / tpb.x, (in_m + tpb.y - 1) / tpb.y);
	
	relu<<<bpg, tpb>>>(in.device_data(), out.device_data(), in_m, in_n);

	hipError_t err = hipGetLastError();
	if(err != hipSuccess){
		std::cerr << "CUDA ERR: " << hipGetErrorString(err) << std::endl;
		throw std::runtime_error("forward kernel failed");
	}
}

template<typename T>
__global__ void tanh(const T* in, T* out, const size_t m, const size_t n){
	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;

	if(row < m && col < n){
		T val = in[row * n + col];
		T numerator = (exp(val) - exp(-1.0 * val));
		T denominator = (exp(val) + exp(-1.0 * val));

		out[row * n + col] = numerator / denominator;
	}
}



template<typename T>
void tanhCall(const Tensor<T>& in, Tensor<T>& out){
	const size_t in_m = in.shape()[0];
	const size_t in_n = in.shape()[1];
	
	dim3 tpb(16,16);
	dim3 bpg((in_n + tpb.x - 1) / tpb.x, (in_m + tpb.y - 1) / tpb.y);
	
	tanh<<<bpg, tpb>>>(in.device_data(), out.device_data(), in_m, in_n);

	hipError_t err = hipGetLastError();
	if(err != hipSuccess){
		std::cerr << "CUDA ERR: " << hipGetErrorString(err) << std::endl;
		throw std::runtime_error("forward kernel failed");
	}
}
template<typename T>
__global__ void softmax(const T* in, T* out , const size_t m, const size_t n){
	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	if(row < m && col < n){
                float max_val = in[row * n];
                for(int i = 1; i < n; i++){
                        max_val = max(max_val, in[row * n + i]);
                }
                float divisor = 0.0f;
                for(int i = 0; i < n; i++){
                        divisor += exp(in[row * n + i] - max_val);
                }
                out[row * n + col] = exp(in[row * n + col] - max_val) / divisor;
        }
}

template<typename T>
void softmaxCall(const Tensor<T>& in, Tensor<T>& out){
	const size_t m = in.shape()[0];
	const size_t n = in.shape()[1];

	dim3 tpb(16, 16);
	dim3 bpg((tpb.x + n - 1) / tpb.x, (tpb.y + m - 1) / tpb.y);

	softmax<<<bpg, tpb>>>(in.device_data(), out.device_data(), m, n);
	hipError_t err = hipGetLastError();
	if(err != hipSuccess){
		throw std::runtime_error("softmax kernel failed.");
	}
}
